#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>

#include "win-gettimeofday.h"

#define ELEMENTS 10000000
#define RANGE 1000
#define NUM_THREADS 1024
#define min(a,b) (a<b?a:b)


// Returns an integer representing the threads global index
//
// This function calculates the global index of the current thread
__device__ int getIdx()
{
	return blockIdx.x * blockDim.x + threadIdx.x;
}

// Takes in a pointer to the array that needs checking
// Returns a boolean indicating whether or not the array is sorted.
//
// Function will iterate over the array and for element in the array it will 
// check that it is not less thatn the element before it. If it is, then it will return false
// and if it makes it to the end of the array, it will return true.
bool succesfullySorted(float* sorted)
{
	for (int i = 1; i < ELEMENTS; i++)
		if (sorted[i] < sorted[i - 1])
			return false;
	return true;
}

// Takes in a pointer to an array
// Returns nothing, bur modifies the array that was passed in.
//
// This function iterates over the array that has been passed in up until the ELEMENTS limit which is defined at the top of this file.
// For each element in the array it will insert a random float value between and the number defined in RANGE.
void genRandomArray(float* arr)
{
	printf("Generating array \n");
	for (int i = 0; i < ELEMENTS; i++)
	{
		arr[i] = ((float)rand() / (float)RAND_MAX) * RANGE;	//Gen random floating point numbers up to 10
	}
	printf("Array Made \n\n");
}

// Takes in two array pointers and two integer values. The pointer d_array is the array that needs to be sorted, and the pointer d_temp
// is the array that the values are stored in after sorting. The integer value width is the current width of the sorted arrays and numSegments
// is the number of segments the array is currently split into.
// Returns nothing, but modifies d_temp. 
//
// This function will assign each thread 2 arrays to merge, and then that thread will merge them into one larger sorted array in d_temp.
// 
// This function does not copy the merge results back to d_array from d_temp. The pointers to the arrays need to be swapped before launching the next kernel.
__global__
void mergeCuda(float* d_array, float* d_temp, int width, int numSegments)
{
	int globalIndex = getIdx();

	if (globalIndex > numSegments)
		return;

	int start = globalIndex * width;
	int end = min(start + width - 1, ELEMENTS - 1);
	int mid = min(start + (width / 2) - 1, ELEMENTS - 1);

	int startPoint = start;
	int midPoint = mid + 1;
	int index = start;

	while (startPoint <= mid && midPoint <= end)
	{
		if (d_array[startPoint] < d_array[midPoint])
			d_temp[index++] = d_array[startPoint++];
		else
			d_temp[index++] = d_array[midPoint++];
	}
	while (startPoint <= mid)
		d_temp[index++] = d_array[startPoint++];
	while (midPoint <= end)
		d_temp[index++] = d_array[midPoint++];
}

// Takes in a pointer to the array that needs to be sorted.
// Returns nothing, but modifies the array that has been passed in.
//
// This function acts as a wrapper for all of the cuda specific parts of the merge sort, such as 
// allocating/copying memory and freeing the memory on the device. As well as determing grid and block sizes and launching the kernels.
void cudaMergeSort(float* arr)
{
	int arrayMemorySize = ELEMENTS * sizeof(float);
	float* d_temp, * d_array;
	hipMalloc(&d_temp, arrayMemorySize);
	hipMalloc(&d_array, arrayMemorySize);
	hipMemcpy(d_array, arr, arrayMemorySize, hipMemcpyHostToDevice);

	for (int width = 2; width < 2 * ELEMENTS; width *= 2)
	{
		int numSegments = ceil((double)ELEMENTS / (double)width);
		int numBlocks = ceil((double)numSegments / (double)NUM_THREADS);
		dim3 gridDim(numBlocks, 1, 1);
		dim3 dimBlock(NUM_THREADS, 1, 1);
		mergeCuda << <gridDim, dimBlock >> > (d_array, d_temp, width, numSegments);

		float* swap = d_array;
		d_array = d_temp;
		d_temp = swap;

	}

	hipMemcpy(arr, d_array, arrayMemorySize, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipFree(d_temp);
	hipFree(d_array);
}

int main()
{


	float* h_array;
	h_array = (float*)malloc(ELEMENTS * sizeof(float));
	genRandomArray(h_array);

	//Print the initial array
	//for (int i = 0; i < ELEMENTS; i++)
	//	printf("%f \n", h_array[i]);

	//Beginning of timed section
	float timeStart, timeEnd, timeTotal;
	timeStart = get_current_time();
	cudaMergeSort(h_array);
	timeEnd = get_current_time();
	timeTotal = timeEnd - timeStart;
	//Ending of timed section

	//Print the array
	//for (int i = 0; i < ELEMENTS; i++)
	//	printf("%d: %f \n", i, h_array[i]);

	printf("Time taken: %f \n", timeTotal);

	if (succesfullySorted(h_array))
		printf("Good sort \n");
	else
		printf("Bad sort \n");

	hipDeviceSynchronize();
	free(h_array);
	return 0;
}
