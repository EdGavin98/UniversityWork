#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
#include "win-gettimeofday.h"

#define ELEMENTS 100000
#define RANGE 1000
#define NUM_THREADS 1024
#define TILE_SIZE 4096
#define min(a,b) (a < b ? a : b)
#define max(a,b) (a > b ? a : b)

// Takes in a pointer to the array the needs checking.
// Returns an integer value indicating if it has been sorted or not (-1 if sorted, if not sorted it returns the index of the first unsorted element).
//
// Function will iterate over the array and for each element in the array it will
// check that it is not less than the element before it. If it is, then it will return false
// and if it makes it to the end of the array then it will return true.
int succesfullySorted(float* sorted)
{
	for (int i = 1; i < ELEMENTS; i++)
	{
		if (sorted[i] < sorted[i - 1])
			return i;
	}
	return -1;
}

// Takes in a pointer to an array
// Returns nothing, but modifies the array that was passed in.
//
// This function iterates over the array that has been passed in up till the ELEMENTS limit, defined at the top of this file.
// For each element in the array it will insert a random float value between 0 and the number defined in RANGE.
void genRandomArray(float* arr)
{
	printf("Generating array \n");
	for (int i = 0; i < ELEMENTS; i++)
	{
		arr[i] = ((float)rand() / (float)RAND_MAX) * RANGE;	//Gen random floating point numbers up to RANGE
	}
	printf("Array Made \n\n");
}


// Takes in three pointers to arrays and two integer values. Pointers A and B are the two arrays that need to
// merged together and C is the array that they shall be merged in to. Integers m and n are the lengths of 
// A and B respectively, length of C is assumed to be m + n.
//
// Returns nothing, but modifies C.
//
// This function performs a merge operation on arrays A and B, and merges them into C.
__device__
void merge(const float* A, int m, const float* B, int n, float* C) {
	int aStart = 0;
	int bStart = 0;
	int outputIndex = 0;
	while ((aStart < m) && (bStart < n))
	{
		if (A[aStart] <= B[bStart])
			C[outputIndex++] = A[aStart++];
		else
			C[outputIndex++] = B[bStart++];
	}
	if (aStart == m)
		for (; bStart < n; bStart++)
			C[outputIndex++] = B[bStart];
	else
		for (; aStart < m; aStart++)
			C[outputIndex++] = A[aStart];
}

// Takes in 3 integers and 2 array pointers. A and B are the arrays that need to be searched, and integers m 
// and n are their lengths. Integer i is the index at which the current thread is going to begin or end it's output.
// Returns an integer 'j' specifying the co rank of one of the elements in the array.
//
// This function uses the input index provide to work out the co ranks of this index in arrays A and B using a binary search.
// It will then return the indexs co rank in array A, and the co rank in array B can then be worked out by doing i - j;
__device__
int coRank(int i, float* A, int m, float* B, int n)
{
	int j = min(i, m);
	int k = i - j;
	int j_low = max(0, i - n);
	int k_low;
	int delta;
	while (true)
	{
		if (j > 0 && k < n && A[j - 1] > B[k])
		{
			delta = ((j - j_low + 1) >> 1);
			k_low = k;
			j = j - delta;
			k = k + delta;
		}
		else if (k > 0 && j < m && B[k - 1] >= A[j])
		{
			delta = ((k - k_low + 1) >> 1);
			j_low = j;
			j = j + delta;
			k = k - delta;
		}
		else
		{
			break;
		}
	}
	return j;
}

// Takes in 3 array pointers, A, B and C and two integers representing the length of A and B (Length of C is m+n)
// Returns nothing, but modifies the array C.
//
// This function will use the co_rank function to divide up the array amongst all of the threads within the block.
// It will then call the merge function so that each thread can merge its section of the array into the output array C.
__device__
void coRankMerge(float* A, int m, float* B, int n, float* C)
{
	int sectionSize = ceil((double)(m + n) / blockDim.x); 
	int outputStart = min(threadIdx.x * sectionSize, m + n);
	int outputEnd = min((threadIdx.x + 1) * sectionSize, m + n);

	int aThreadStart = coRank(outputStart, A, m, B, n);
	int aThreadEnd = coRank(outputEnd, A, m, B, n);

	int bThreadStart = outputStart - aThreadStart;
	int bThreadEnd = outputEnd - aThreadEnd;

	merge(&A[aThreadStart], aThreadEnd - aThreadStart, &B[bThreadStart], bThreadEnd - bThreadStart, &C[outputStart]);
}

// Takes in 2 array pointers d_array and d_temp, and the current width of the sorted arrays.
// Returns nothing, but modifies the arrays that are passed in.
//
// This function will determine the start and end index of each pair of arrays that needs to be merged and assign it to a thread block
// it will then call the coRankMerge function so that they can begin the merge operation.
// This function does not copy the sorted arrays back to d_array, it will be necessary to swap the pointers before the next kernel launch.
__global__
void mergeCuda(float* d_array, float* d_temp, int width)
{
	int start = blockIdx.x * width;
	int end = min(start + width - 1, ELEMENTS - 1);
	int mid = min(start + (width >> 1), ELEMENTS - 1);

	coRankMerge(&d_array[start], mid - start, &d_array[mid], end - mid + 1, &d_temp[start]);
}


// Takes in three array pointers for the Input arrays A and B, and the output array C. As well as 
// integer values m and n, representing the length of A and B respectively. 
// Returns nothing, but modifies the output array C.
//
// This function uses the co ranking algorithm to split up the array into blocks that can fit inside
// shared memory, it will then split up each of those tiles among the threads in the block so that they can be merged.
__device__
void coRankTiledMerge(float* A, int m, float* B, int n, float* C, int blocksPerArray)
{
	__shared__ float sharedAB[TILE_SIZE << 1];
	float* sharedA = &sharedAB[0];
	float* sharedB = &sharedAB[TILE_SIZE];

	int segSize = ceil(((double)(m + n) / blocksPerArray));

	int outBlockStart = min((blockIdx.x % blocksPerArray) * segSize, m + n);
	int outBlockEnd = min(((blockIdx.x % blocksPerArray) + 1) * segSize, m + n);

	//Calculate the co rank values for the whole block
	//Only need one thread as it can be put in shared memory
	if (threadIdx.x == 0)
	{
		sharedA[0] = coRank(outBlockStart, A, m, B, n);
		sharedA[1] = coRank(outBlockEnd, A, m, B, n);
	}

	__syncthreads();
													 
	int aBlockStart = sharedA[0];							 
	int aBlockEnd = sharedA[1];								 
																 
	int bBlockStart = outBlockStart - aBlockStart;			 
	int bBlockEnd = outBlockEnd - aBlockEnd;				 
																 
	__syncthreads();											 
																 
	int aChunkSize = aBlockEnd - aBlockStart;			 
	int bChunkSize = bBlockEnd - bBlockStart;					 
	int cChunkSize = outBlockEnd - outBlockStart;				

	//Load values into shared memory
	for (int i = 0; i < TILE_SIZE; i += blockDim.x) //Increase i by blockDim.x so all values are next to each other when being loaded, allowing for memory coalescing
	{
		int index = i + threadIdx.x;
		if (index < aChunkSize)
			sharedA[index] = A[aBlockStart + index];
		if (index < bChunkSize)
			sharedB[index] = B[bBlockStart + index];
	}

	__syncthreads();

	int outputThreadStart = min(threadIdx.x * (TILE_SIZE / blockDim.x), cChunkSize);
	int outputThreadEnd = min((threadIdx.x + 1) * (TILE_SIZE / blockDim.x), cChunkSize);

	int aThreadStart = coRank(outputThreadStart, sharedA, min(TILE_SIZE, aChunkSize), sharedB, min(TILE_SIZE, bChunkSize));
	int aThreadEnd = coRank(outputThreadEnd, sharedA, min(TILE_SIZE, aChunkSize), sharedB, min(TILE_SIZE, bChunkSize));

	int bThreadStart = outputThreadStart - aThreadStart;
	int bThreadEnd = outputThreadEnd - aThreadEnd;

	merge(&sharedA[aThreadStart], aThreadEnd - aThreadStart, &sharedB[bThreadStart], bThreadEnd - bThreadStart, &C[outBlockStart + outputThreadStart]);

}

// Takes in two array pointers, d_array and d_temp, the integer value width, representing the current width of the sorted arrays
// and also the integer value blocksPerArray, representing the number of blocks that will be tiled across the arrays.
// Returns nothing, but modifies the array d_temp.
//
// This function will determine the start and end index for each array pair that needs to be merged, and will then assign each block to
// an array depending on its index.
// 
// This function does not copy the sorted arrays back to d_array, it is necessary to swap the pointers before the next kernel launch.
__global__
void mergeCudaTiled(float* d_array, float* d_temp, int width, int blocksPerArray)
{
	int arrayNumber = blockIdx.x / blocksPerArray;
	int start = arrayNumber * width;
	int end = min(start + width - 1, ELEMENTS - 1);
	int mid = min(start + (width / 2) - 1, ELEMENTS - 1);

	coRankTiledMerge(&d_array[start], (mid - start) + 1, &d_array[mid + 1], end - mid, &d_temp[start], blocksPerArray);
}

// Takes in the array that needs to be sorted
// Returns nothing, but modifies the array that was passed in.
//
// This function acts as a wrapper for all of the cuda functionality, such as allocating, copying and freeing memory on the device, as well as
// handling all of block and grid size calculations and launching the kernel.
void cudaMergeSortTiled(float* arr)
{
	int arrayMemorySize = ELEMENTS * sizeof(float);
	float* d_temp, * d_array;
	hipMalloc(&d_temp, arrayMemorySize);
	hipMalloc(&d_array, arrayMemorySize);
	hipMemcpy(d_array, arr, arrayMemorySize, hipMemcpyHostToDevice);

	for (int width = 2; width < 2 * ELEMENTS; width *= 2)
	{
		int numBlocks = 0;
		int numBlocksPerArray = 1;

		if (width > TILE_SIZE)
		{
			int numSegments = ceil((double)ELEMENTS / width);
			numBlocksPerArray = width / TILE_SIZE;
			numBlocks = numBlocksPerArray * numSegments;
		}
		else
		{
			numBlocks = ceil((double)ELEMENTS / (double)width);
		}

		int threads = min(width, NUM_THREADS);

		dim3 gridDim(numBlocks, 1, 1);
		dim3 blockDim(threads, 1, 1);

		//Time spent allocating and storing into shared memory is long when there are large numbers of arrays as each needs at least 1 block
		//so use global for the first few iterations
		if (width <= 128)
			mergeCuda<<<gridDim, blockDim>>>(d_array, d_temp, width);
		else
			mergeCudaTiled<<<gridDim, blockDim >>>(d_array, d_temp, width, numBlocksPerArray);

		float* swap = d_array;
		d_array = d_temp;
		d_temp = swap;
	}
	hipMemcpy(arr, d_array, arrayMemorySize, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(d_temp);
	hipFree(d_array);
}

// Takes in the array that needs to be sorted
// Returns nothing, but modifies the array that was passed in.
//
// This function acts as a wrapper for all of the cuda functionality, such as allocating, copying and freeing memory on the device, as well as
// handling all of block and grid size calculations and launching the kernel.
void cudaMergeSort(float* arr)
{
	int arrayMemorySize = ELEMENTS * sizeof(float);
	float* d_temp, * d_array;
	hipMalloc(&d_temp, arrayMemorySize);
	hipMalloc(&d_array, arrayMemorySize);
	hipMemcpy(d_array, arr, arrayMemorySize, hipMemcpyHostToDevice);

	for (int width = 2; width < 2 * ELEMENTS; width *= 2)
	{
		
		int numBlocks = ceil((double)ELEMENTS / (double)width);
		int threads = min(width, NUM_THREADS);
		dim3 gridDim(numBlocks, 1, 1);
		dim3 blockDim(threads, 1, 1);

		mergeCuda << <gridDim, blockDim >> > (d_array, d_temp, width);

		float* swap = d_array;
		d_array = d_temp;
		d_temp = swap;

	}
	hipMemcpy(arr, d_array, arrayMemorySize, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(d_temp);
	hipFree(d_array);
}

int main()
{
	float* h_array;
	h_array = (float*)malloc(ELEMENTS * sizeof(float));
	genRandomArray(h_array);

	//Print the initial array
	//for (int i = 0; i < ELEMENTS; i++)
	//	printf("%f \n", h_array[i]);

	//Beginning of timed section
	float timeStart, timeEnd, timeTotal;
	timeStart = get_current_time();
	cudaMergeSortTiled(h_array); //Sort with tiling
	//cudaMergeSort(h_array);    //Sort without tiling
	timeEnd = get_current_time();
	timeTotal = timeEnd - timeStart;
	printf("Time taken: %f \n", timeTotal);
	//End of timed section

	//Print the sorted array
	//for (int i = 0; i < ELEMENTS; i++)
	//	printf("%d: %f \n", i, h_array[i]);

	int isSorted = succesfullySorted(h_array);
	if (isSorted == -1)
		printf("Good sort \n");
	else
		printf("Bad sort: %d \n", isSorted);

	free(h_array);
	
	return 0;
}